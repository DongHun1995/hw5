#include "hip/hip_runtime.h"
#include <cstdio>
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

#define TS 64
#define WPT 8
#define RTS TS / WPT


static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K)
{  
  int globalRow = blockDim.y * blockIdx.y + threadIdx.y;
  int globalCol = WPT * blockDim.x * blockIdx.x + threadIdx.x;
  int row = threadIdx.y;
  int col = threadIdx.x;

  __shared__ float Asub[TS][TS];
  __shared__ float Bsub[TS][TS];

  float acc[WPT];
  for (int i =0; i < WPT; i++)
  {
    acc[i] = 0.0;
  }

  for (int offset =0; offset < K; offset += TS)
  {
    int tiledRow = offset + row;
    int tiledCol = offset + col;

    for (int i=0; i < WPT; i++)
    {
      Asub[row][col + i * RTS] = A[globalRow * K + (tiledCol + i * RTS)];
      Bsub[row][col + i * RTS] = B[tiledRow * N + (globalCol + i * RTS)];
    }

    __syncthreads();

    for (int k=0; k < TS; ++k)
    {
      for (int i =0; i < WPT; i++)
      {
        acc[i] += Asub[row][k] * Bsub[k][col + i * RTS];
      }
    }
    __syncthreads();
  }
  for (int i =0; i < WPT; i++)
  {
    C[globalRow * N + (globalCol + i * RTS)] = acc[i];
  }
  
}

#define NGPU 4
#define EVENTS_PER_GPU 1 //INCREASE as needed

static size_t Mbegin[NGPU], Mend[NGPU];
static size_t ngpu;
static hipStream_t streams[NGPU];
static hipEvent_t events[NGPU][EVENTS_PER_GPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];

void matmul_initialize(int M, int N, int K) 
{
  ngpu = 4;

  for (size_t i = 0; i < ngpu; i++)
  {
    Mbegin[i] = M / ngpu * i;
    Mend[i] = M / ngpu * (i + 1);
    if (i == ngpu - 1) Mend[i] = M;
  }

  for (size_t i = 0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
    for (int j=0; j < EVENTS_PER_GPU; j++)
    {
      CHECK_CUDA(hipEventCreate(&events[i][j]));
    }
  }

  for (size_t i =0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMalloc(&A_gpu[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }

}


void matmul(const float *A, const float *B, float *C, int M, int N, int K) 
{
  for (size_t i =0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &A[Mbegin[i] * K], (Mend[i] - Mbegin[i]) * K * sizeof(float), hipMemcpyHostToDevice, streams[i]));
    CHECK_CUDA(hipMemcpyAsync(B_gpu[i], B, K * N * sizeof(float), hipMemcpyHostToDevice, streams[i]));
  }

  for (size_t i =0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    dim3 blockDim(TS / WPT, TS);
    dim3 gridDim((N + TS - 1) / TS, (Mend[i] - Mbegin[i] + TS - 1) / TS);
    matmul_kernel<<<gridDim, blockDim>>>(A_gpu[i], B_gpu[i], C_gpu[i], Mend[i] - Mbegin[i], N, K);
    CHECK_CUDA(hipGetLastError());
  }

  for(size_t i =0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(&C[Mbegin[i] * N], C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float), hipMemcpyDeviceToHost, streams[i]));
  }

  for (size_t i =0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamSynchronize(streams[i]));
  }
}


void matmul_finalize() 
{
  for(size_t i = 0; i < ngpu; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
    for (int j = 0; j < EVENTS_PER_GPU; j++)
    {
      CHECK_CUDA(hipEventDestroy(events[i][j]));
    }
  }
}