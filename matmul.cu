#include "hip/hip_runtime.h"
#include <cstdio>
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j >= N) return;
  float sum = 0.0;
  for (int k = 0; k < K; k++)
  {
    sum += A[i * K + k] * B[k * N + j];
  }
  C[i * N + j] = sum;
}

#define BLOCKS 4

static size_t Mbegin[BLOCKS], Mend[BLOCKS];
static hipStream_t data_stream, calc_stream;
static hipEvent_t events[BLOCKS];
static float *A_gpu, *B_gpu, *C_gpu;

void matmul_initialize(int M, int N, int K) 
{
  for (size_t i = 0; i < BLOCKS; i++)
  {
    Mbegin[i] = M / BLOCKS * i;
    Mend[i] = M / BLOCKS * (i + 1);
    if (i == BLOCKS - 1) Mend[i] = M;
  }

  CHECK_CUDA(hipStreamCreate(&data_stream));
  CHECK_CUDA(hipStreamCreate(&calc_stream));
  for (int i = 0; i < BLOCKS; i++)
  {
    CHECK_CUDA(hipEventCreate(&events[i]));
  }
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));
}


void matmul(const float *A, const float *B, float *C, int M, int N, int K) 
{
  CHECK_CUDA(hipMemcpyAsync(B_gpu, B, K * N * sizeof(float), hipMemcpyHostToDevice, data_stream));
  for(int i = 0; i < BLOCKS; i++)
  {
    CHECK_CUDA(hipMemcpyAsync(&A_gpu[Mbegin[i] * K], &A[Mbegin[i] * K], (Mend[i] - Mbegin[i]) * K * sizeof(float), hipMemcpyHostToDevice, data_stream));
    CHECK_CUDA(hipEventRecord(events[i], data_stream));
  }

  for (int i =0; i < BLOCKS; i++)
  {
    dim3 blockDim(32, 32);
    dim3 gridDim((Mend[i] - Mbegin[i] + 32 - 1) / 32, (N + 32 - 1) / 32);
    CHECK_CUDA(hipStreamWaitEvent(calc_stream, events[i]));
    matmul_kernel<<<gridDim, blockDim, 0, calc_stream>>>(&A_gpu[Mbegin[i] * K], B_gpu, &C_gpu[Mbegin[i] * N], (Mend[i] - Mbegin[i]), N, K);
  }

  CHECK_CUDA(hipStreamSynchronize(calc_stream));
  CHECK_CUDA(hipMemcpyAsync(C, C_gpu, M * N * sizeof(float), hipMemcpyDeviceToHost, data_stream));
  CHECK_CUDA(hipStreamSynchronize(data_stream));

}


void matmul_finalize() 
{
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
  CHECK_CUDA(hipStreamDestroy(data_stream));
  CHECK_CUDA(hipStreamDestroy(calc_stream));
  for (int i =0; i < BLOCKS; i++)
  {
    CHECK_CUDA(hipEventDestroy(events[i]));
  }
}
